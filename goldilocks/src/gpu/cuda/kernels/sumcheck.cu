#include "hip/hip_runtime.h"
#include "../includes/goldilocks/fp_impl.cuh"
#include "../includes/goldilocks/fp2_impl.cuh"

using namespace goldilocks;

// TODO
__device__ fp2 combine_function(fp2* evals, unsigned int start, unsigned int stride, unsigned int num_args) {
    fp2 result = fp2::one();
    for (int i = 0; i < num_args; i++) result *= evals[start + i * stride];
    return result;
}

extern "C" __global__ void combine(fp2* buf, unsigned int size, unsigned int num_args) {
    const int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    while (idx < size) {
        buf[idx] = combine_function(buf, idx, size, num_args);
        idx += blockDim.x * gridDim.x;
    }
}

extern "C" __global__ void sum(fp2* data, fp2* result, unsigned int stride, unsigned int index) {
    const int tid = threadIdx.x;
    for (unsigned int s = stride; s > 0; s >>= 1) {
        int idx = tid;
        while (idx < s) {
            data[idx] += data[idx + s];
            idx += blockDim.x;
        }
        __syncthreads();
    }
    if (tid == 0) result[index] = data[0];
}

extern "C" __global__ void fold_into_half(
    unsigned int num_vars, unsigned int initial_poly_size, unsigned int num_blocks_per_poly, fp2* polys, fp2* buf, fp* challenge
) {
    int tid = (blockIdx.x % num_blocks_per_poly) * blockDim.x + threadIdx.x;
    const int stride = 1 << (num_vars - 1);
    const int buf_offset = (blockIdx.x / num_blocks_per_poly) * stride;
    const int poly_offset = (blockIdx.x / num_blocks_per_poly) * initial_poly_size;
    while (tid < stride) {
        if (*challenge == fp::zero()) buf[buf_offset + tid] = polys[poly_offset + tid];
        else if (*challenge == fp::one()) buf[buf_offset + tid] = polys[poly_offset + tid + stride];
        else buf[buf_offset + tid] = (polys[poly_offset + tid + stride] - polys[poly_offset + tid]).scalar_mul(*challenge) + polys[poly_offset + tid];
        tid += blockDim.x * num_blocks_per_poly;
    }
}

extern "C" __global__ void fold_into_half_in_place(
    unsigned int num_vars, unsigned int initial_poly_size, unsigned int num_blocks_per_poly, fp2* polys, fp2* challenge
) {
    int tid = (blockIdx.x % num_blocks_per_poly) * blockDim.x + threadIdx.x;
    const int stride = 1 << (num_vars - 1);
    const int offset = (blockIdx.x / num_blocks_per_poly) * initial_poly_size;
    while (tid < stride) {
        int idx = offset + tid;
        polys[idx] = (*challenge) * (polys[idx + stride] - polys[idx]) + polys[idx];
        tid += blockDim.x * num_blocks_per_poly;
    }
}

// TODO : Pass transcript and squeeze random challenge using hash function
extern "C" __global__ void squeeze_challenge(fp2* challenges, unsigned int index) {
    if (threadIdx.x == 0) {
        challenges[index] = fp2(fp(1034));
    }
}
